
#include <hip/hip_runtime.h>
// Tests that "sm_XX" gets correctly converted to "compute_YY" when we invoke
// fatbinary.

// RUN: %clang -### --target=x86_64-linux-gnu -c --cuda-gpu-arch=sm_20 --cuda-path=%S/Inputs/CUDA_80/usr/local/cuda %s 2>&1 \
// RUN: | FileCheck -check-prefixes=CUDA,SM20 %s
// RUN: %clang -### --target=x86_64-linux-gnu -c --cuda-gpu-arch=sm_21 --cuda-path=%S/Inputs/CUDA_80/usr/local/cuda %s 2>&1 \
// RUN: | FileCheck -check-prefixes=CUDA,SM21 %s
// RUN: %clang -### --target=x86_64-linux-gnu -c --cuda-gpu-arch=sm_30 --cuda-path=%S/Inputs/CUDA_80/usr/local/cuda %s 2>&1 \
// RUN: | FileCheck -check-prefixes=CUDA,SM30 %s
// RUN: %clang -### --target=x86_64-linux-gnu -c --cuda-gpu-arch=sm_32 --cuda-path=%S/Inputs/CUDA_80/usr/local/cuda %s 2>&1 \
// RUN: | FileCheck -check-prefixes=CUDA,SM32 %s
// RUN: %clang -### --target=x86_64-linux-gnu -c --cuda-gpu-arch=sm_35 --cuda-path=%S/Inputs/CUDA_80/usr/local/cuda %s 2>&1 \
// RUN: | FileCheck -check-prefixes=CUDA,SM35 %s
// RUN: %clang -### --target=x86_64-linux-gnu -c --cuda-gpu-arch=sm_37 --cuda-path=%S/Inputs/CUDA_80/usr/local/cuda %s 2>&1 \
// RUN: | FileCheck -check-prefixes=CUDA,SM37 %s
// RUN: %clang -### --target=x86_64-linux-gnu -c --cuda-gpu-arch=sm_50 --cuda-path=%S/Inputs/CUDA_80/usr/local/cuda %s 2>&1 \
// RUN: | FileCheck -check-prefixes=CUDA,SM50 %s
// RUN: %clang -### --target=x86_64-linux-gnu -c --cuda-gpu-arch=sm_52 --cuda-path=%S/Inputs/CUDA_80/usr/local/cuda %s 2>&1 \
// RUN: | FileCheck -check-prefixes=CUDA,SM52 %s
// RUN: %clang -### --target=x86_64-linux-gnu -c --cuda-gpu-arch=sm_53 --cuda-path=%S/Inputs/CUDA_80/usr/local/cuda %s 2>&1 \
// RUN: | FileCheck -check-prefixes=CUDA,SM53 %s
// RUN: %clang -### --target=x86_64-linux-gnu -c --cuda-gpu-arch=sm_60 --cuda-path=%S/Inputs/CUDA_80/usr/local/cuda %s 2>&1 \
// RUN: | FileCheck -check-prefixes=CUDA,SM60 %s
// RUN: %clang -### --target=x86_64-linux-gnu -c --cuda-gpu-arch=sm_61 --cuda-path=%S/Inputs/CUDA_80/usr/local/cuda %s 2>&1 \
// RUN: | FileCheck -check-prefixes=CUDA,SM61 %s
// RUN: %clang -### --target=x86_64-linux-gnu -c --cuda-gpu-arch=sm_62 --cuda-path=%S/Inputs/CUDA_80/usr/local/cuda %s 2>&1 \
// RUN: | FileCheck -check-prefixes=CUDA,SM62 %s
// RUN: %clang -### --target=x86_64-linux-gnu -c --cuda-gpu-arch=sm_70 --cuda-path=%S/Inputs/CUDA_111/usr/local/cuda %s 2>&1 \
// RUN: | FileCheck -check-prefixes=CUDA,SM70 %s
// RUN: %clang -x hip -### --target=x86_64-linux-gnu -c --cuda-gpu-arch=gfx600 -nogpuinc -nogpulib %s 2>&1 \
// RUN: | FileCheck -check-prefixes=HIP,GFX600 %s
// RUN: %clang -x hip -### --target=x86_64-linux-gnu -c --cuda-gpu-arch=gfx601 -nogpuinc -nogpulib %s 2>&1 \
// RUN: | FileCheck -check-prefixes=HIP,GFX601 %s
// RUN: %clang -x hip -### --target=x86_64-linux-gnu -c --cuda-gpu-arch=gfx602 -nogpuinc -nogpulib %s 2>&1 \
// RUN: | FileCheck -check-prefixes=HIP,GFX602 %s
// RUN: %clang -x hip -### --target=x86_64-linux-gnu -c --cuda-gpu-arch=gfx700 -nogpuinc -nogpulib %s 2>&1 \
// RUN: | FileCheck -check-prefixes=HIP,GFX700 %s
// RUN: %clang -x hip -### --target=x86_64-linux-gnu -c --cuda-gpu-arch=gfx701 -nogpuinc -nogpulib %s 2>&1 \
// RUN: | FileCheck -check-prefixes=HIP,GFX701 %s
// RUN: %clang -x hip -### --target=x86_64-linux-gnu -c --cuda-gpu-arch=gfx702 -nogpuinc -nogpulib %s 2>&1 \
// RUN: | FileCheck -check-prefixes=HIP,GFX702 %s
// RUN: %clang -x hip -### --target=x86_64-linux-gnu -c --cuda-gpu-arch=gfx703 -nogpuinc -nogpulib %s 2>&1 \
// RUN: | FileCheck -check-prefixes=HIP,GFX703 %s
// RUN: %clang -x hip -### --target=x86_64-linux-gnu -c --cuda-gpu-arch=gfx704 -nogpuinc -nogpulib %s 2>&1 \
// RUN: | FileCheck -check-prefixes=HIP,GFX704 %s
// RUN: %clang -x hip -### --target=x86_64-linux-gnu -c --cuda-gpu-arch=gfx705 -nogpuinc -nogpulib %s 2>&1 \
// RUN: | FileCheck -check-prefixes=HIP,GFX705 %s
// RUN: %clang -x hip -### --target=x86_64-linux-gnu -c --cuda-gpu-arch=gfx801 -nogpuinc -nogpulib %s 2>&1 \
// RUN: | FileCheck -check-prefixes=HIP,GFX801 %s
// RUN: %clang -x hip -### --target=x86_64-linux-gnu -c --cuda-gpu-arch=gfx802 -nogpuinc -nogpulib %s 2>&1 \
// RUN: | FileCheck -check-prefixes=HIP,GFX802 %s
// RUN: %clang -x hip -### --target=x86_64-linux-gnu -c --cuda-gpu-arch=gfx803 -nogpuinc -nogpulib %s 2>&1 \
// RUN: | FileCheck -check-prefixes=HIP,GFX803 %s
// RUN: %clang -x hip -### --target=x86_64-linux-gnu -c --cuda-gpu-arch=gfx805 -nogpuinc -nogpulib %s 2>&1 \
// RUN: | FileCheck -check-prefixes=HIP,GFX805 %s
// RUN: %clang -x hip -### --target=x86_64-linux-gnu -c --cuda-gpu-arch=gfx810 -nogpuinc -nogpulib %s 2>&1 \
// RUN: | FileCheck -check-prefixes=HIP,GFX810 %s
// RUN: %clang -x hip -### --target=x86_64-linux-gnu -c --cuda-gpu-arch=gfx900 -nogpuinc -nogpulib %s 2>&1 \
// RUN: | FileCheck -check-prefixes=HIP,GFX900 %s
// RUN: %clang -x hip -### --target=x86_64-linux-gnu -c --cuda-gpu-arch=gfx902 -nogpuinc -nogpulib %s 2>&1 \
// RUN: | FileCheck -check-prefixes=HIP,GFX902 %s
// RUN: %clang -x hip -### --target=x86_64-linux-gnu -c --cuda-gpu-arch=amdgcnspirv -nogpuinc -nogpulib %s 2>&1 \
// RUN: | FileCheck -check-prefixes=HIP,SPIRV %s

// CUDA: ptxas
// CUDA-SAME: -m64
// CUDA: fatbinary

// HIP: clang-offload-bundler

// SM20:--image=profile=sm_20{{.*}}
// SM21:--image=profile=sm_21{{.*}}
// SM30:--image=profile=sm_30{{.*}}
// SM32:--image=profile=sm_32{{.*}}
// SM35:--image=profile=sm_35{{.*}}
// SM37:--image=profile=sm_37{{.*}}
// SM50:--image=profile=sm_50{{.*}}
// SM52:--image=profile=sm_52{{.*}}
// SM53:--image=profile=sm_53{{.*}}
// SM60:--image=profile=sm_60{{.*}}
// SM61:--image=profile=sm_61{{.*}}
// SM62:--image=profile=sm_62{{.*}}
// SM70:--image=profile=sm_70{{.*}}
// GFX600:-targets=host-x86_64-unknown-linux-gnu,hipv4-amdgcn-amd-amdhsa--gfx600
// GFX601:-targets=host-x86_64-unknown-linux-gnu,hipv4-amdgcn-amd-amdhsa--gfx601
// GFX602:-targets=host-x86_64-unknown-linux-gnu,hipv4-amdgcn-amd-amdhsa--gfx602
// GFX700:-targets=host-x86_64-unknown-linux-gnu,hipv4-amdgcn-amd-amdhsa--gfx700
// GFX701:-targets=host-x86_64-unknown-linux-gnu,hipv4-amdgcn-amd-amdhsa--gfx701
// GFX702:-targets=host-x86_64-unknown-linux-gnu,hipv4-amdgcn-amd-amdhsa--gfx702
// GFX703:-targets=host-x86_64-unknown-linux-gnu,hipv4-amdgcn-amd-amdhsa--gfx703
// GFX704:-targets=host-x86_64-unknown-linux-gnu,hipv4-amdgcn-amd-amdhsa--gfx704
// GFX705:-targets=host-x86_64-unknown-linux-gnu,hipv4-amdgcn-amd-amdhsa--gfx705
// GFX801:-targets=host-x86_64-unknown-linux-gnu,hipv4-amdgcn-amd-amdhsa--gfx801
// GFX802:-targets=host-x86_64-unknown-linux-gnu,hipv4-amdgcn-amd-amdhsa--gfx802
// GFX803:-targets=host-x86_64-unknown-linux-gnu,hipv4-amdgcn-amd-amdhsa--gfx803
// GFX805:-targets=host-x86_64-unknown-linux-gnu,hipv4-amdgcn-amd-amdhsa--gfx805
// GFX810:-targets=host-x86_64-unknown-linux-gnu,hipv4-amdgcn-amd-amdhsa--gfx810
// GFX900:-targets=host-x86_64-unknown-linux-gnu,hipv4-amdgcn-amd-amdhsa--gfx900
// GFX902:-targets=host-x86_64-unknown-linux-gnu,hipv4-amdgcn-amd-amdhsa--gfx902
// SPIRV:-targets=host-x86_64-unknown-linux-gnu,hip-spirv64-amd-amdhsa--amdgcnspirv
